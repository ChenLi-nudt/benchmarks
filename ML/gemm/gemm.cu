#include "hip/hip_runtime.h"
#define BLOCK_SIZE 16
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include <stdlib.h>

__global__ void
matrixMulCUDA(float *C, const float *A, const float *B, int wA, int wB, int hA, int tran = 0, int tranA = 0, int tranB = 0, float alpha = 1, float beta = 0)
{

	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;
	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Index of the first sub-matrix of A processed by the block

	int aBegin = wA * BLOCK_SIZE * by;

	// Index of the last sub-matrix of A processed by the block

	int aEnd = aBegin + wA - 1;

	// Step size used to iterate through the sub-matrices of A

	int aStep = BLOCK_SIZE;

	// Index of the first sub-matrix of B processed by the block

	int bBegin = BLOCK_SIZE * bx;

	// Step size used to iterate through the sub-matrices of B

	int bStep = BLOCK_SIZE * wB;

	// Csub is used to store the element of the block sub-matrix

	// that is computed by the thread

	float Csub = 0;

	// Loop over all the sub-matrices of A and B

	// required to compute the block sub-matrix
	int i = 0;
	for (int a = aBegin, b = bBegin;

		a <= aEnd;

	a += aStep, b += bStep)
	{

		// Declaration of the shared memory array As used to

		// store the sub-matrix of A

		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

		// Declaration of the shared memory array Bs used to

		// store the sub-matrix of B

		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from device memory

		// to shared memory; each thread loads

		// one element of each matrix
		//if (by*BLOCK_SIZE + ty < hA && bx*BLOCK_SIZE + tx < wB){
		int gid = a + wA * ty + tx;
		if (gid < wA*hA && i*BLOCK_SIZE + tx < wA){
			if (tranA == 0)
			{
				As[ty][tx] = A[gid];
			}
			else
			{

				int rid = gid / wA;
				int cid = gid % wA;
				int target_id = cid*hA + rid;
				As[ty][tx] = A[target_id];
			}
		}
		gid = b + wB * ty + tx;
		if (gid < wA*wB ){
			if (tranB == 0)
			{
				Bs[ty][tx] = B[gid];
			}
			else
			{

				int rid = gid / wB;
				int cid = gid % wB;
				int target_id = cid*wA + rid;
				Bs[ty][tx] = B[target_id];
			}
		}

		//}

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices together;

		// each thread computes one element

		// of the block sub-matrix


		if (by*BLOCK_SIZE + ty < hA && bx*BLOCK_SIZE + tx < wB){
#pragma unroll
			for (int k = 0; k < BLOCK_SIZE; ++k)
			{
				if (i*BLOCK_SIZE + k < wA){
					Csub += As[ty][k] * Bs[k][tx];
				}
			}
		}

		// Synchronize to make sure that the preceding

		// computation is done before loading two new

		// sub-matrices of A and B in the next iteration

		__syncthreads();
		i++;
	}

	// Write the block sub-matrix to device memory;

	// each thread writes one element

	int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	if (by*BLOCK_SIZE + ty < hA && bx*BLOCK_SIZE + tx < wB){
		if (tran == 0)
		{
			C[c + wB * ty + tx] = C[c + wB * ty + tx] * beta + Csub*alpha;
		}
		else
		{
			int gid = c + wB * ty + tx;
			int rid = gid / wB;
			int cid = gid % wB;
			int target_id = cid*hA + rid;
			C[target_id] = C[target_id] * beta + Csub*alpha;
		}
	}
}

float *cuda_make_array(float *x, size_t n)
{
    float *x_gpu;
    size_t size = sizeof(float)*n;
    hipError_t status = hipMalloc((void **)&x_gpu, size);
    //check_error(status);
    if(x){
        status = hipMemcpy(x_gpu, x, size, hipMemcpyHostToDevice);
        //check_error(status);
    }
    if(!x_gpu) printf("Cuda malloc failed\n");
    return x_gpu;
}



int main(){

    int wA = 512;
    int wB = 512;
    int hA = 512;

    float *Acpu = (float*)malloc(hA*wA*sizeof(float));
    float *Bcpu = (float*)malloc(wA*wB*sizeof(float));
    float *Ccpu = (float*)malloc(hA*wB*sizeof(float));

    // initalize A and B

    float *A = cuda_make_array(Acpu, hA*wA);
    float *B = cuda_make_array(Bcpu, wA*wB);
    float *C = cuda_make_array(Ccpu, hA*wB);

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((wB + threads.x - 1) / threads.x, (hA + threads.y - 1) / threads.y);
    matrixMulCUDA<< < grid, threads >> >(C, A, B, wA, wB, hA, 0, 0, 0, 1, 0);

    // check results with a cpu matrix multiplication

    free(Acpu);
    free(Bcpu);
    free(Ccpu);

    hipFree(A);
    hipFree(B);
    hipFree(C);        
    return 0;
}
