#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "mycublas.h"
__global__ void transpose(float *AT, const float *A, int rows, int cols){

	int bx = blockIdx.x;
	int tx = threadIdx.x;
	int gid = bx*blockDim.x + tx;
	if (gid < rows*cols)
	{
		int rid = gid / cols;
		int cid = gid % cols;
		int target_id = cid*rows + rid;
		int btx = target_id / blockDim.x;
		int ttx = target_id % blockDim.x;
		AT[btx*blockDim.x + ttx] = A[gid];
	}
}

template <int BLOCK_SIZE> __global__ void

matrixMulCUDA(float *C, const float *A, const float *B, int wA, int wB, int hA)

{

	// Block index

	int bx = blockIdx.x;

	int by = blockIdx.y;



	// Thread index

	int tx = threadIdx.x;

	int ty = threadIdx.y;

	/*if (by*BLOCK_SIZE + ty >= hA)
	{
	return;
	}

	if (bx*BLOCK_SIZE + tx >= wB)
	{
	return;
	}*/

	// Index of the first sub-matrix of A processed by the block

	int aBegin = wA * BLOCK_SIZE * by;



	// Index of the last sub-matrix of A processed by the block

	int aEnd = aBegin + wA - 1;



	// Step size used to iterate through the sub-matrices of A

	int aStep = BLOCK_SIZE;



	// Index of the first sub-matrix of B processed by the block

	int bBegin = BLOCK_SIZE * bx;



	// Step size used to iterate through the sub-matrices of B

	int bStep = BLOCK_SIZE * wB;



	// Csub is used to store the element of the block sub-matrix

	// that is computed by the thread

	float Csub = 0;



	// Loop over all the sub-matrices of A and B

	// required to compute the block sub-matrix

	for (int a = aBegin, b = bBegin;

		a <= aEnd;

	a += aStep, b += bStep)

	{



		// Declaration of the shared memory array As used to

		// store the sub-matrix of A

		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];



		// Declaration of the shared memory array Bs used to

		// store the sub-matrix of B

		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];



		// Load the matrices from device memory

		// to shared memory; each thread loads

		// one element of each matrix
		if (by*BLOCK_SIZE + ty < hA && bx*BLOCK_SIZE + tx < wB){
			As[ty][tx] = A[a + wA * ty + tx];

			Bs[ty][tx] = B[b + wB * ty + tx];
		}


		// Synchronize to make sure the matrices are loaded

		__syncthreads();



		// Multiply the two matrices together;

		// each thread computes one element

		// of the block sub-matrix

#pragma unroll



		for (int k = 0; k < BLOCK_SIZE; ++k)

		{
			if (k < wA && by*BLOCK_SIZE + ty < hA && bx*BLOCK_SIZE + tx < wB){
				Csub += As[ty][k] * Bs[k][tx];
			}

		}



		// Synchronize to make sure that the preceding

		// computation is done before loading two new

		// sub-matrices of A and B in the next iteration

		__syncthreads();

	}



	// Write the block sub-matrix to device memory;

	// each thread writes one element

	int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	if (by*BLOCK_SIZE + ty < hA && bx*BLOCK_SIZE + tx < wB)
		C[c + wB * ty + tx] = Csub;

}
int mycublasSgemm(int transa, int transb, int m, int n, int k,
	const float *alpha, const float *A, int lda, const float *B, int ldb,
	const float *beta, float *C, int ldc)
{

	if (transa == 1 && transb == 1){
		dim3 threads(16, 16);

		dim3 grid((n + threads.x - 1) / threads.x, (m + threads.y - 1) / threads.y);
		//printf("grid.x %d grid.y %d\n", grid.x, grid.y);
		float *CT = cuda_make_array(0, m*n);
		matrixMulCUDA<16> << < grid, threads >> >(CT, A, B, k, n, m);
		int block_dim = 16;
		int grid_dim = (m*n + block_dim - 1) / block_dim;
		transpose<<<grid_dim,block_dim>>>(C, CT, m, n);
		cuda_free(CT);

	}
	else if (transa == 1 && transb == 0)
	{
		float *BT = cuda_make_array(0, k*n);
		int block_dim = 16;
		int grid_dim = (k*n + block_dim - 1) / block_dim;
		transpose << <grid_dim, block_dim >> >(BT, B, n, k);

		dim3 threads(16, 16);
		dim3 grid((n + threads.x - 1) / threads.x, (m + threads.y - 1) / threads.y);
		//printf("grid.x %d grid.y %d\n", grid.x, grid.y);
		float *CT = cuda_make_array(0, m*n);
		matrixMulCUDA<16> << < grid, threads >> >(CT, A, BT, k, n, m);

		block_dim = 16;
		grid_dim = (m*n + block_dim - 1) / block_dim;
		transpose << <grid_dim, block_dim >> >(C, CT, m, n);
		cuda_free(CT);
		cuda_free(BT);
	}

	else if (transa == 0 && transb == 1)
	{
		float *AT = cuda_make_array(0, k*m);
		int block_dim = 16;
		int grid_dim = (k*m + block_dim - 1) / block_dim;
		transpose << <grid_dim, block_dim >> >(AT, A, k, m);

		dim3 threads(16, 16);
		dim3 grid((n + threads.x - 1) / threads.x, (m + threads.y - 1) / threads.y);
		//printf("grid.x %d grid.y %d\n", grid.x, grid.y);
		float *CT = cuda_make_array(0, m*n);
		matrixMulCUDA<16> << < grid, threads >> >(CT, AT, B, k, n, m);

		block_dim = 16;
		grid_dim = (m*n + block_dim - 1) / block_dim;
		transpose << <grid_dim, block_dim >> >(C, CT, m, n);
		cuda_free(CT);
		cuda_free(AT);
	}

	else
	{
		float *AT = cuda_make_array(0, k*m);
		int block_dim = 16;
		int grid_dim = (k*m + block_dim - 1) / block_dim;
		transpose << <grid_dim, block_dim >> >(AT, A, k, m);

		float *BT = cuda_make_array(0, k*n);
		block_dim = 16;
		grid_dim = (k*n + block_dim - 1) / block_dim;
		transpose << <grid_dim, block_dim >> >(BT, B, n, k);

		dim3 threads(16, 16);
		dim3 grid((n + threads.x - 1) / threads.x, (m + threads.y - 1) / threads.y);
		//printf("grid.x %d grid.y %d\n", grid.x, grid.y);
		float *CT = cuda_make_array(0, m*n);
		matrixMulCUDA<16> << < grid, threads >> >(CT, AT, BT, k, n, m);

		block_dim = 16;
		grid_dim = (m*n + block_dim - 1) / block_dim;
		transpose << <grid_dim, block_dim >> >(C, CT, m, n);
		cuda_free(CT);
		cuda_free(AT);
		cuda_free(BT);
	}


	return 0;

}
